// Teebone's first CUDA programming

#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel function in GPU
__global__ void square_num(float* a, int n)
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if(id < n)
        a[id] = a[id]*a[id];
}

#define N 10
#define block_size 4

//main routine
int main(){
    float *a_host, *a_cuda;
    a_host = (float*)malloc(N*sizeof(float)); // float number in host(CPU).
    hipMalloc((void **) &a_cuda, N*sizeof(float)); //float number in device(GPU)
    
    for(int i=0;i<N;i++)
        a_host[i] = (float)i;

    hipMemcpy(a_cuda,a_host,N*sizeof(float),hipMemcpyHostToDevice);
    int num_blocks = N/block_size + (N%block_size == 0 ? 0:1);
    // do calculation on GPU
    square_num <<< num_blocks, block_size >>>(a_cuda, N);
    //retrive value from GPU device
    hipMemcpy(a_host,a_cuda,N*sizeof(float),hipMemcpyDeviceToHost);
    //print result
    for(int i=0;i<N;i++)
        printf("a[%d] = %f\n",i,a_host[i]);

    free(a_host);
    hipFree(a_cuda);
    return 0;

}
