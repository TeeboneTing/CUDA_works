// CUDA programming: factorial problem


#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
// kernel function in GPU
__global__ void findFac( int* a,int* idx ,int n)
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if(id < n){
        //printf("id=%d  %d mod %d == %d\n",id,(*a),b[id],(*a)%b[id]);
        int b;
        id == 0 ? b=2 : b= 1+2*id;
        if((*a)%b == 0)
            *idx = b;
    }
}

//#define N 1
#define block 100

//main routine
int main(int argc, char** argv){
    //check usage
    if (argc != 3){
        printf("usage: ./main [input file] [output file]\n");
        return 1;
    }
    
    int a_host,id_host;
    double tmp;
    int  *a_cuda, *id_cuda; //indicates which b_cuda[id] factorial
    FILE* input = fopen(argv[1],"r");
    fscanf(input,"%d",&a_host);
    fclose(input);
    
    tmp = ceil(sqrt(a_host))/2;

    hipMalloc((void **) &a_cuda, sizeof(int)) ; 
    hipMemcpy(a_cuda,&a_host,sizeof(int),hipMemcpyHostToDevice) ;
    hipMalloc((void **) &id_cuda, sizeof(int)) ; 

    //int block_size = int(tmp)/block + (int(tmp)%block == 0 ? 0:1);
    // do calculation on GPU
    //findFac <<< block, block_size >>>(a_cuda,id_cuda, tmp);
    findFac <<< 1, tmp >>>(a_cuda,id_cuda, tmp);
    //retrive value from GPU device
    hipMemcpy(&id_host,id_cuda,sizeof(int),hipMemcpyDeviceToHost);

    hipFree(a_cuda);
    hipFree(id_cuda);

    //output result to file
    FILE* output = fopen(argv[2],"w");
    fprintf(output,"%d\n%d\n",a_host/id_host,id_host);
    fclose(output);
    return 0;

}
